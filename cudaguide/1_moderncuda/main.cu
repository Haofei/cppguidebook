#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvfunctional>
#include "cudapp.cuh" // 小彭老师现代 CUDA 框架，更符合现代 C++ 风格，减少官方 C 风格接口的繁琐

using namespace cudapp;

__global__ void kernel(int x) {
    printf("内核参数 x = %d\n", x);
    printf("线程编号 (%d, %d)\n", blockIdx.x, threadIdx.x);
}

int main() {
    // 启动内核的3种方式
    // 1. 官方三箭头语法糖（常用）
    // kernel<<<blockDim, gridDim, dynamicSmemBytes, stream>>>(...)
    int x = 42;
    kernel<<<3, 4, 0, 0>>>(x);

    // 2. hipLaunchKernel
    void *args[] = {&x};
    CHECK_CUDA(hipLaunchKernel(reinterpret_cast<const void*>(kernel), dim3(3), dim3(4), args, 0, 0));

    // 3. cudaLaunchKernelEx
    hipLaunchConfig_t cfg{};
    cfg.blockDim = dim3(3); // threadIdx 的变化范围
    cfg.gridDim = dim3(4); // blockIdx 的变化范围
    cfg.dynamicSmemBytes = 0; // shared-memory 大小（暂不使用）
    cfg.stream = 0; // 在 0 号流（默认流）上启动
    cfg.attrs = nullptr;
    cfg.numAttrs = 0;
    CHECK_CUDA(cudaLaunchKernelEx(&cfg, kernel, x));

    // 还能查询内核名字
    const char *name;
    CHECK_CUDA(cudaFuncGetName(&name, kernel));
    printf("内核名字：%s\n", name);

    // 1. 强制同步：等待此前启动过的所有内核执行完成
    CHECK_CUDA(hipDeviceSynchronize());

    // 2. 仅同步 0 号流（默认流）：等待此前在 0 号流上启动过的所有内核执行完成
    CHECK_CUDA(hipStreamSynchronize(0));

    // 3. 仅同步 0 号流，但使用小彭老师现代 CUDA 框架
    CudaStream::defaultStream().join();

    return 0;
}
